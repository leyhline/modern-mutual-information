/**
 * Copyright 2017 Thomas Leyh <leyht@informatik.uni-freiburg.de>
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <iterator>

#include "CudaMI.h"
#include "utilities.h"

// TODO Check return codes for errors!!!

CudaMI::CudaMI(const int shift_from, const int shift_to,
			   const int binsX, const int binsY,
		       const float minX, const float maxX,
			   const float minY, const float maxY,
		       const float* const dataX,
			   const float* const dataY, const size_t data_size)
	: data_size(data_size),
	  result_size(shift_to - shift_from),
	  calculation_done(false)
{
	h_result = (float*)malloc((size_t)result_size * sizeof(float));
	hipMalloc((void**)&d_X, data_size * sizeof(int));
	hipMalloc((void**)&d_Y, data_size * sizeof(int));
}

CudaMI::~CudaMI()
{
	free(h_result);
	hipFree(d_X);
	hipFree(d_Y);
}

const float* CudaMI::shifted_mutual_information()
{
	if (!calculation_done)
	{
		// TODO Code comes here.
		calculation_done = true;
	}
	return h_result;
}

int CudaMI::getSizeOfShiftedArray() const
{
	return result_size;
}
