/**
 * Copyright 2017 Thomas Leyh <leyht@informatik.uni-freiburg.de>
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <iterator>

#include "CudaMI.h"

// TODO Check return codes for errors!!!

CudaMI::CudaMI(const int shift_from, const int shift_to,
			   const unsigned int binsX, const unsigned int binsY,
		       const float minX, const float maxX,
			   const float minY, const float maxY,
		       const float* const dataX,
			   const float* const dataY, const size_t data_size)
	: byte_size(data_size * sizeof(float)), result_size(shift_to - shift_from)
{
	h_result = (float*)malloc(result_size * sizeof(float));
	hipMalloc((void**)&d_X, byte_size);
	hipMalloc((void**)&d_Y, byte_size);
	hipMemcpy(d_X, dataX, byte_size, hipMemcpyHostToDevice);
	hipMemcpy(d_Y, dataY, byte_size, hipMemcpyHostToDevice);
}

CudaMI::~CudaMI()
{
	free(h_result);
	hipFree(d_X);
	hipFree(d_Y);
}

const float* CudaMI::shifted_mutual_information()
{
	// TODO Code comes here.
	return h_result;
}

int CudaMI::getSizeOfShiftedArray() const
{
	return result_size;
}
