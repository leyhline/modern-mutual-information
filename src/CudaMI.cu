#include "hip/hip_runtime.h"
/**
* Copyright 2018, University of Freiburg
* Optophysiology Lab.
* Thomas Leyh <thomas.leyh@mailbox.org>
*
* Licensed under the Apache License, Version 2.0 (the "License");
* you may not use this file except in compliance with the License.
* You may obtain a copy of the License at
*
*     http://www.apache.org/licenses/LICENSE-2.0
*
* Unless required by applicable law or agreed to in writing, software
* distributed under the License is distributed on an "AS IS" BASIS,
* WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
* See the License for the specific language governing permissions and
* limitations under the License.
*/

#include <iterator>

#include "CudaMI.h"
#include "utilities.h"

/// Calculate shifted mutual information.
/// Every kernel calculates one shift.
__global__ void shifted_mutual_information_kernel(
		const int shift_from, const int shift_to,
		const float minX, const float maxX,
		const float minY, const float maxY,
		const int* indicesX, const int* indicesY, int data_size,
		float* d_result)
{
	int index = blockDim.x * blockIdx.x + threadIdx.x;
	int result_size = shift_to - shift_from;
	if (index >= result_size)
		return;

	int hist[BINS][BINS];
	int count = 0;

	// Initialize whole histogram with 0.
	for (int x = 0; x < BINS; ++x)
	{
		for (int y = 0; y < BINS; ++y)
		{
			hist[x][y] = 0;
		}
	}

	// This is dynamic indexing and therefore we get into slow local memory. :(
	// Increment 2D histogram according to indices.
	int k_shift_from = shift_from + index;
	if (k_shift_from < 0)
	{
		for (int i = 0; i < (data_size - k_shift_from); ++i)
		{
			int x = indicesX[i];
			int y = indicesY[i + k_shift_from];
			if (x < BINS && y < BINS)
			{
				++hist[x][y];
				++count;
			}
		}
	}
	else
	{
		for (int i = 0; i < (data_size - k_shift_from); ++i)
		{
			int x = indicesX[i + k_shift_from];
			int y = indicesY[i];
			if (x < BINS && y < BINS)
			{
				++hist[x][y];
				++count;
			}
		}
	}

	// Calculate 1D histograms.
	int histX[BINS];
	for (int x = 0; x < BINS; ++x)
		histX[x] = 0;
	int histY[BINS];
	for (int y = 0; y < BINS; ++y)
		histY[y] = 0;
	for (int x = 0; x < BINS; ++x)
	{
		for (int y = 0; y < BINS; ++y)
		{
			histX[x] += hist[x][y];
			histY[y] += hist[x][y];
		}
	}

	// Calculate mutual information.
	float mi = 0;
	for (int x = 0; x < BINS; ++x)
	{
		for (int y = 0; y < BINS; ++y)
		{
			if (hist[x][y] > 0)
			{
				float p_xy = (float)(hist[x][y]) / count;
				float p_x =  (float)(histX[x]) / count;
				float p_y =  (float)(histY[y]) / count;
				mi += p_xy * log2(p_xy / (p_x * p_y));
			}
		}
	}

	d_result[index] = mi;
}

/// Calculate shifted mutual information.
/// Every kernel calculates one shift. Each kernel writes the results
/// of the histogram computation not to local but to fast shared memory.
__global__ void shifted_mutual_information_kernel_sharedmem(
		const int shift_from, const int shift_to,
		const float minX, const float maxX,
		const float minY, const float maxY,
		const int* indicesX, const int* indicesY, int data_size,
		float* d_result)
{
	int index = blockDim.x * blockIdx.x + threadIdx.x;
	int tid = threadIdx.x;
	int result_size = shift_to - shift_from;
	if (index >= result_size)
		return;

	__shared__ int hist[BINS][BINS][CudaMI::block_size];
	int count = 0;

	// Initialize whole histogram with 0.
	for (int x = 0; x < BINS; ++x)
	{
		for (int y = 0; y < BINS; ++y)
		{
			hist[x][y][tid] = 0;
		}
	}

	// Increment 2D histogram according to indices.
	// This in now in shared (instead of local) memory and faster.
	int k_shift_from = shift_from + index;
	if (k_shift_from < 0)
	{
		for (int i = 0; i < (data_size - k_shift_from); ++i)
		{
			int x = indicesX[i];
			int y = indicesY[i + k_shift_from];
			if (x < BINS && y < BINS)
			{
				++hist[x][y][tid];
				++count;
			}
		}
	}
	else
	{
		for (int i = 0; i < (data_size - k_shift_from); ++i)
		{
			int x = indicesX[i + k_shift_from];
			int y = indicesY[i];
			if (x < BINS && y < BINS)
			{
				++hist[x][y][tid];
				++count;
			}
		}
	}

	// Calculate 1D histograms.
	int histX[BINS];
	for (int x = 0; x < BINS; ++x)
		histX[x] = 0;
	int histY[BINS];
	for (int y = 0; y < BINS; ++y)
		histY[y] = 0;
	for (int x = 0; x < BINS; ++x)
	{
		for (int y = 0; y < BINS; ++y)
		{
			histX[x] += hist[x][y][tid];
			histY[y] += hist[x][y][tid];
		}
	}

	// Calculate mutual information.
	float mi = 0;
	for (int x = 0; x < BINS; ++x)
	{
		for (int y = 0; y < BINS; ++y)
		{
			if (hist[x][y][tid] > 0)
			{
				float p_xy = (float)(hist[x][y][tid]) / count;
				float p_x =  (float)(histX[x]) / count;
				float p_y =  (float)(histY[y]) / count;
				mi += p_xy * log2(p_xy / (p_x * p_y));
			}
		}
	}

	d_result[index] = mi;
}

// TODO Check return codes for errors!!!
// TODO Check if arguments are logical.

CudaMI::CudaMI(const int shift_from, const int shift_to,
		       const float minX, const float maxX,
			   const float minY, const float maxY,
		       const float* const dataX,
			   const float* const dataY, const int data_size)
	: shift_from(shift_from), shift_to(shift_to),
	  minX(minX), maxX(maxX),
	  minY(minY), maxY(maxY),
	  data_size(data_size), result_size(shift_to - shift_from), calculation_done(false)
{
	check_constructor();
	h_result = (float*)malloc(result_size * sizeof(float));
	hipMalloc((void**)&d_result, result_size * sizeof(float));
	hipMalloc((void**)&d_X, data_size * sizeof(int));
	hipMalloc((void**)&d_Y, data_size * sizeof(int));
	// This can be done on the GPU later, too.
	auto indicesX = calculate_indices_1d(BINS, minX, maxX, dataX, dataX + data_size);
	auto indicesY = calculate_indices_1d(BINS, minY, maxY, dataY, dataY + data_size);
	hipMemcpy(d_X, indicesX.data(), data_size * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_Y, indicesY.data(), data_size * sizeof(int), hipMemcpyHostToDevice);
}

CudaMI::~CudaMI()
{
	free(h_result);
	hipFree(d_result);
	hipFree(d_X);
	hipFree(d_Y);
}

const float* CudaMI::shifted_mutual_information()
{
	if (!calculation_done)
	{
		// Ceiling of integer division result_size / block_size.
		int grid_size = ((result_size - 1) / block_size) + 1;
		shifted_mutual_information_kernel_sharedmem<<<grid_size, block_size>>>(
				shift_from, shift_to, minX, maxX, minY, maxY, d_X, d_Y, data_size, d_result);
		hipMemcpy(h_result, d_result, result_size * sizeof(float), hipMemcpyDeviceToHost);
		calculation_done = true;
	}
	return h_result;
}

int CudaMI::getSizeOfShiftedArray() const
{
	return result_size;
}

void CudaMI::check_constructor() const
{
	if (shift_from >= shift_to)
		throw std::logic_error("shift_from has to be smaller than shift_to.");
}
